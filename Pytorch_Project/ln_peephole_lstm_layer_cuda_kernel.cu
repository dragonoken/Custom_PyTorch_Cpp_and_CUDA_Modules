#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t const &z)
{
	return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t tanh(scalar_t const &z)
{
	const auto exp_n2z = exp(-2 * z);
	return (1.0 - exp_n2z) / (1.0 + exp_n2z);
}
/*
template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t const &z)
{
	const auto exp_nz = exp(-z);
	return exp_nz / pow((1.0 + exp_nz), 2);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_tanh(scalar_t const &z)
{
	const auto exp_n2z = exp(-2 * z);
	return 4 * exp_n2z / pow((1.0 + exp_n2z), 2);
}
*/
template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid_with_output(scalar_t const &a)
{
	return a * (1.0 - a);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_tanh_with_output(scalar_t const &a)
{
	return 1.0 - (a * a);
}

template <typename scalar_t>
__global__ void forward_part_0(
	const scalar_t* __restrict__ hidden,
	const scalar_t* __restrict__ cell,
	scalar_t* __restrict__ hiddens_storage,
	scalar_t* __restrict__ cells_storage,
	scalar_t* __restrict__ current_gate,
	const scalar_t* __restrict__ mean_fig,
	const scalar_t* __restrict__ var_fig,
	const scalar_t epsilon,
	scalar_t* __restrict__ stds_storage,
	scalar_t* __restrict__ normalized_storage,
	const scalar_t* __restrict__ gamma_fig,
	const scalar_t* __restrict__ bias_fig,
	scalar_t* __restrict__ activated_storage,
	scalar_t* __restrict__ forgotten_cell,
	const scalar_t* __restrict__ dropout_candidate_cell,
	const int64_t batch_size,
	const int64_t state_size,
	const int64_t state_size_3)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int process_idx = blockIdx.z;
			if (process_idx < 4)
			{
				if (process_idx < 3) //Normalizations and stuff
				{
					const int mean_var_std_idx = batch * 3 + process_idx;
					const scalar_t std = sqrt(var_fig[mean_var_std_idx] + epsilon);
					if (column == 0)
					{
						stds_storage[mean_var_std_idx] = std;
					}
					const int gate_val_storage_idx = batch * state_size_3 + process_idx * state_size + column;
					const int gate_val_local_idx = gate_val_storage_idx + batch * state_size;
					scalar_t gate_val = (current_gate[gate_val_local_idx] - mean_fig[mean_var_std_idx]) / std;
					normalized_storage[gate_val_storage_idx] = gate_val;
					const int gamma_bias_idx = process_idx * state_size + column;
					if (process_idx < 2) //forget gate & input gate
					{
						gate_val = sigmoid((gate_val * gamma_fig[gamma_bias_idx]) + bias_fig[gamma_bias_idx]);
						if (process_idx == 0) //forget cell memory
						{
							const int local_state_idx = batch * state_size + column;
							forgotten_cell[local_state_idx] = gate_val * cell[local_state_idx];
						}
						activated_storage[gate_val_storage_idx] = gate_val;
						current_gate[gate_val_local_idx] = gate_val;
					}
					else //candidate cell
					{
						gate_val = tanh((gate_val * gamma_fig[gamma_bias_idx]) + bias_fig[gamma_bias_idx]);
						activated_storage[gate_val_storage_idx] = gate_val;
						current_gate[gate_val_local_idx] = gate_val * dropout_candidate_cell[batch * state_size + column];
					}
				}
				else //Hidden, Cell Storage
				{
					const int local_state_idx = batch * state_size + column;
					hiddens_storage[local_state_idx] = hidden[local_state_idx];
					cells_storage[local_state_idx] = cell[local_state_idx];
				}
			}
		}
	}
}

template <typename scalar_t>
__global__ void forward_part_1(
	const scalar_t* __restrict__ forgotten_cell,
	const scalar_t* __restrict__ current_gate,
	scalar_t* __restrict__ cell,
	const int64_t batch_size,
	const int64_t state_size,
	const int64_t gate_size)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int local_state_idx = batch * state_size + column;
			const int local_input_gate_idx = batch * gate_size + state_size + column;
			cell[local_state_idx] = forgotten_cell[local_state_idx] + current_gate[local_input_gate_idx] * current_gate[local_input_gate_idx + state_size];
		}
	}
}

template <typename scalar_t>
__global__ void forward_part_2(
	scalar_t* __restrict__ cell,
	const scalar_t* __restrict__ mean,
	const scalar_t* __restrict__ var,
	const scalar_t epsilon,
	scalar_t* __restrict__ new_cell_stds_storage,
	scalar_t* __restrict__ new_cell_normalized_storage,
	const scalar_t* __restrict__ gamma_new_cell,
	const scalar_t* __restrict__ beta_new_cell,
	scalar_t* __restrict__ hc,
	scalar_t* __restrict__ current_gate,
	const scalar_t* __restrict__ weight_co,
	const int64_t batch_size,
	const int64_t state_size,
	const int64_t state_size_2)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int cell_idx = batch * state_size + column;
			const scalar_t std = sqrt(var[batch] + epsilon);
			if (column == 0)
			{
				new_cell_stds_storage[batch] = std;
			}
			scalar_t cell_val = (cell[cell_idx] - mean[batch]) / std;
			new_cell_normalized_storage[cell_idx] = cell_val;
			cell_val = (cell_val * gamma_new_cell[column]) + beta_new_cell[column];
			cell[cell_idx] = cell_val;
			const int hc_idx = cell_idx + (batch + 1) * state_size;
			hc[hc_idx] = cell_val;
			current_gate[hc_idx + (batch + 1) * state_size_2] += cell_val * weight_co[column];
		}
	}
}

template <typename scalar_t>
__global__ void forward_part_3(
	const scalar_t* __restrict__ current_gate,
	const scalar_t* __restrict__ mean,
	const scalar_t* __restrict__ var,
	const scalar_t epsilon,
	scalar_t* __restrict__ output_gate_stds_storage,
	scalar_t* __restrict__ output_gate_normalized_storage,
	const scalar_t* __restrict__ gamma_o,
	const scalar_t* __restrict__ bias_o,
	scalar_t* __restrict__ output_gate_activated_storage,
	const scalar_t* __restrict__ cell,
	scalar_t* __restrict__ tanh_new_cell_storage,
	scalar_t* __restrict__ hidden,
	scalar_t* __restrict__ hc,
	scalar_t* __restrict__ outputs,
	const scalar_t* __restrict__ dropout_output,
	const int64_t batch_size,
	const int64_t state_size,
	const int64_t state_size_3)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int state_and_output_gate_storage_idx = batch * state_size + column;
			const int hc_idx = state_and_output_gate_storage_idx + batch * state_size;
			const int output_gate_idx = state_and_output_gate_storage_idx + (batch + 1) * state_size_3;
			const scalar_t std = sqrt(var[batch] + epsilon);
			if (column == 0)
			{
				output_gate_stds_storage[batch] = std;
			}
			scalar_t output_gate_val = (current_gate[output_gate_idx] - mean[batch]) / std;
			output_gate_normalized_storage[state_and_output_gate_storage_idx] = output_gate_val;
			output_gate_val = sigmoid((output_gate_val * gamma_o[column]) + bias_o[column]);
			output_gate_activated_storage[state_and_output_gate_storage_idx] = output_gate_val;
			const scalar_t tanh_cell = tanh(cell[state_and_output_gate_storage_idx]);
			tanh_new_cell_storage[state_and_output_gate_storage_idx] = tanh_cell;
			const scalar_t hidden_val = output_gate_val * tanh_cell;
			hidden[state_and_output_gate_storage_idx] = hidden_val;
			hc[hc_idx] = hidden_val;
			outputs[state_and_output_gate_storage_idx] = hidden_val * dropout_output[state_and_output_gate_storage_idx];
		}
	}
}

std::vector<at::Tensor> ln_peephole_lstm_layer_cuda_forward(
	at::Tensor const &input,
	at::Tensor const &weight_ih,
	at::Tensor const &weight_hh,
	at::Tensor const &weight_ch,
	at::Tensor const &bias,
	at::Tensor const &gamma_f,
	at::Tensor const &gamma_i,
	at::Tensor const &gamma_g,
	at::Tensor const &gamma_o,
	at::Tensor const &gamma_new_cell,
	at::Tensor const &beta_new_cell,
	at::Tensor &hidden,
	at::Tensor &cell,
	double const &epsilon,
	double const &dropout_p,
	bool const &dropout_on_output,
	bool const &training,
	int64_t const &sequence_length,
	int64_t const &batch_size,
	int64_t const &input_size,
	int64_t const &state_size,
	int64_t const &state_size_2,
	int64_t const &state_size_3,
	int64_t const &gate_size)
{
	const auto options = weight_ih.options();

	auto hiddens = at::empty({ sequence_length, batch_size, state_size }, options);
	auto cells = at::empty({ sequence_length + 1, batch_size, state_size }, options);

	auto gates_fig_stds = at::empty({ sequence_length, batch_size, 3, 1 }, options);
	auto gates_fig_normalized = at::empty({ sequence_length, batch_size, 3, state_size }, options);
	auto gates_fig = at::empty({ sequence_length, batch_size, 3, state_size }, options);

	auto gates_o_stds = at::empty({ sequence_length, batch_size, 1 }, options);
	auto gates_o_normalized = at::empty({ sequence_length, batch_size, state_size }, options);
	auto gates_o = at::empty({ sequence_length, batch_size, state_size }, options);

	auto new_cells_stds = at::empty({ sequence_length, batch_size, 1 }, options);
	auto new_cells_normalized = at::empty({ sequence_length, batch_size, state_size }, options);

	auto tanh_new_cells = at::empty({ sequence_length, batch_size, state_size }, options);

	auto outputs = at::empty({ sequence_length, batch_size, state_size }, options);

	at::Tensor dropout;
	if (dropout_p <= 0. || !training) { dropout = at::ones({ 2, sequence_length, batch_size, state_size }, options); }
	else
	{
		if (dropout_p >= 1.) { dropout = at::zeros({ 2, sequence_length, batch_size, state_size }, options); }
		else { dropout = at::bernoulli(at::zeros({ 2, sequence_length, batch_size, state_size }, options), (1 - dropout_p)).div(1 - dropout_p); }

		if (!dropout_on_output) { dropout[1] = 1; }
	}
	const auto dropout_candidate_cell = dropout[0];
	const auto dropout_output = dropout[1];

	const auto ih = at::matmul(input, weight_ih.t());

	auto hc = at::cat({ hidden, cell }, 1);
	const auto weight_hc_h = at::cat({ weight_hh.t(),
									   at::cat({ weight_ch.slice(0, 0, state_size).diag(),
												 weight_ch.slice(0, state_size, state_size_2).diag(),
												 at::zeros({ state_size_2, state_size }, options) }).t() });

	const auto weight_co = weight_ch.slice(0, state_size_2);

	const auto gamma_fig = at::stack({ gamma_f, gamma_i, gamma_g });

	const auto bias_fig = bias.slice(0, 0, state_size_3).view({ 3, state_size });
	const auto bias_o = bias.slice(0, state_size_3);

	at::Tensor current_gate;
	auto forgotten_cell = at::empty_like(cell);
	at::Tensor mean;
	at::Tensor var;

	const dim3 threads(32, 8);
	const dim3 blocks_0((state_size + threads.x - 1) / threads.x,
		                (batch_size + threads.y - 1) / threads.y,
						4);
	const dim3 blocks_1((state_size + threads.x - 1) / threads.x,
		                (batch_size + threads.y - 1) / threads.y);

	AT_DISPATCH_FLOATING_TYPES(ih.type(), "ln_peephole_lstm_layer_cuda_forward", ([&] {
		for (int i = 0; i < sequence_length; i++)
		{
			current_gate = at::addmm(ih[i], hc, weight_hc_h).view({ batch_size, 4, state_size });
			mean = current_gate.slice(1, 0, 3).mean(/*dim=*/2, /*keepdim=*/false);
			var = current_gate.slice(1, 0, 3).var(/*dim=*/2, /*unbiased=*/false, /*keepdim=*/false);

			forward_part_0<scalar_t> <<<blocks_0, threads>>> (
				hidden.data<scalar_t>(),
				cell.data<scalar_t>(),
				hiddens[i].data<scalar_t>(),
				cells[i].data<scalar_t>(),
				current_gate.data<scalar_t>(),
				mean.data<scalar_t>(),
				var.data<scalar_t>(),
				epsilon,
				gates_fig_stds[i].data<scalar_t>(),
				gates_fig_normalized[i].data<scalar_t>(),
				gamma_fig.data<scalar_t>(),
				bias_fig.data<scalar_t>(),
				gates_fig[i].data<scalar_t>(),
				forgotten_cell.data<scalar_t>(),
				dropout_candidate_cell[i].data<scalar_t>(),
				batch_size,
				state_size,
				state_size_3);

			forward_part_1<scalar_t> <<<blocks_1, threads>>> (
				forgotten_cell.data<scalar_t>(),
				current_gate.data<scalar_t>(),
				cell.data<scalar_t>(),
				batch_size,
				state_size,
				gate_size);

			mean = cell.mean(/*dim=*/1, /*keepdim=*/false);
			var = cell.var(/*dim=*/1, /*unbiased=*/false, /*keepdim=*/false);

			forward_part_2<scalar_t> <<<blocks_1, threads>>> (
				cell.data<scalar_t>(),
				mean.data<scalar_t>(),
				var.data<scalar_t>(),
				epsilon,
				new_cells_stds[i].data<scalar_t>(),
				new_cells_normalized[i].data<scalar_t>(),
				gamma_new_cell.data<scalar_t>(),
				beta_new_cell.data<scalar_t>(),
				hc.data<scalar_t>(),
				current_gate.data<scalar_t>(),
				weight_co.data<scalar_t>(),
				batch_size,
				state_size,
				state_size_2);

			mean = current_gate.select(1, 3).mean(/*dim=*/1, /*keepdim=*/false);
			var = current_gate.select(1, 3).var(/*dim=*/1, /*unbiased=*/false, /*keepdim=*/false);

			forward_part_3<scalar_t> <<<blocks_1, threads>>> (
				current_gate.data<scalar_t>(),
				mean.data<scalar_t>(),
				var.data<scalar_t>(),
				epsilon,
				gates_o_stds[i].data<scalar_t>(),
				gates_o_normalized[i].data<scalar_t>(),
				gamma_o.data<scalar_t>(),
				bias_o.data<scalar_t>(),
				gates_o[i].data<scalar_t>(),
				cell.data<scalar_t>(),
				tanh_new_cells[i].data<scalar_t>(),
				hidden.data<scalar_t>(),
				hc.data<scalar_t>(),
				outputs[i].data<scalar_t>(),
				dropout_output[i].data<scalar_t>(),
				batch_size,
				state_size,
				state_size_3);
		}
	}));
	cells[sequence_length] = cell;

	return { outputs,
		hc.slice(1, 0, state_size).contiguous(),
		hc.slice(1, state_size).contiguous(),
		input,
		hiddens,
		cells,
		gates_fig,
		gates_fig_normalized,
		gates_fig_stds,
		gates_o,
		gates_o_normalized,
		gates_o_stds,
		new_cells_normalized,
		new_cells_stds,
		tanh_new_cells,
		dropout };
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void backward_preparation(
	scalar_t* __restrict__ grad_output,
	const scalar_t* __restrict__ dropout_output,
	const scalar_t* __restrict__ dropout_candidate_cell,
	const scalar_t* __restrict__ cells,
	const scalar_t* __restrict__ gates_fig,
	const scalar_t* __restrict__ gates_o,
	scalar_t* __restrict__ grad_gates_layer_normalized,
	scalar_t* __restrict__ gates_fig_stds,
	scalar_t* __restrict__ gates_o_stds,
	scalar_t* __restrict__ new_cells_stds,
	const scalar_t* __restrict__ tanh_new_cells,
	scalar_t* __restrict__ grad_new_cells_wrt_tanh_new_cell,
	scalar_t* __restrict__ forget_gates,
	const size_t n_total_batches,
	const size_t state_size,
	const size_t state_size_2,
	const size_t state_size_3,
	const size_t gate_size)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < n_total_batches)
		{
			const int process_idx = blockIdx.z;
			if (process_idx < 6)
			{
				if (process_idx == 0)
				{
					const int cell_forget_idx = batch * state_size + column;
					const scalar_t forget_gate_val = gates_fig[batch * state_size_3 + column];
					forget_gates[cell_forget_idx] = forget_gate_val;
					grad_gates_layer_normalized[batch * gate_size + column]
						= cells[cell_forget_idx]
						* d_sigmoid_with_output(forget_gate_val);
					if (column == 0)
					{
						gates_fig_stds[batch * 3] *= state_size;
					}
				}
				else{if (process_idx == 1)
				{
					const int dropout_idx = batch * state_size + column;
					const int candidate_cell_idx = dropout_idx + (batch + 1) * state_size_2;
					const int input_gate_idx = candidate_cell_idx - state_size;
					const int store_idx = input_gate_idx + batch * state_size;
					grad_gates_layer_normalized[store_idx] = gates_fig[candidate_cell_idx]
						                                     * d_sigmoid_with_output(gates_fig[input_gate_idx])
						                                     * dropout_candidate_cell[dropout_idx];
					if (column == 0)
					{
						gates_fig_stds[batch * 3 + 1] *= state_size;
					}
				}
				else{if (process_idx == 2)
				{
					const int dropout_idx = batch * state_size + column;
					const int candidate_cell_idx = dropout_idx + (batch + 1) * state_size_2;
					const int input_gate_idx = candidate_cell_idx - state_size;
					const int store_idx = candidate_cell_idx + batch * state_size;
					grad_gates_layer_normalized[store_idx] = gates_fig[input_gate_idx]
						                                     * d_tanh_with_output(gates_fig[candidate_cell_idx])
						                                     * dropout_candidate_cell[dropout_idx];
					if (column == 0)
					{
						gates_fig_stds[batch * 3 + 2] *= state_size;
					}
				}
				else{if (process_idx == 3)
				{
					const int tanh_and_output_idx = batch * state_size + column;
					grad_gates_layer_normalized[batch * gate_size + column + state_size_3]
						= tanh_new_cells[tanh_and_output_idx]
						* d_sigmoid_with_output(gates_o[tanh_and_output_idx]);
					if (column == 0)
					{
						gates_o_stds[batch] *= state_size;
					}
				}
				else{if (process_idx == 4)
				{
					const int index = batch * state_size + column;
					grad_output[index] *= dropout_output[index];
					if (column == 0)
					{
						new_cells_stds[batch] *= state_size;
					}
				}
				else{if (process_idx == 5)
				{
					const int index = batch * state_size + column;
					grad_new_cells_wrt_tanh_new_cell[index] = d_tanh_with_output(tanh_new_cells[index]) * gates_o[index];
				}}}}}}
			}
		}
	}
}

template <typename scalar_t>
__global__ void backward_loop_part_0(
	const scalar_t* __restrict__ grad_hidden,
	scalar_t* __restrict__ grad_new_cell_wrt_tanh_new_cell,
	const scalar_t* __restrict__ grad_output,
	scalar_t* __restrict__ grad_gate_layer_normalized,
	const scalar_t* __restrict__ gamma_o,
	scalar_t* __restrict__ grad_output_gate_normalized,
	const size_t batch_size,
	const size_t state_size,
	const size_t state_size_3)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int local_state_idx = batch * state_size + column;
			const int gate_idx = local_state_idx + (batch + 1) * state_size_3;
			scalar_t grad_val = grad_hidden[local_state_idx] + grad_output[local_state_idx];
			grad_new_cell_wrt_tanh_new_cell[local_state_idx] *= grad_val;
			grad_val *= grad_gate_layer_normalized[gate_idx];
			grad_gate_layer_normalized[gate_idx] = grad_val;
			grad_val *= gamma_o[column];
			grad_output_gate_normalized[local_state_idx] = grad_val;
		}
	}
}

template <typename scalar_t>
__global__ void backward_loop_part_1(
	const scalar_t* __restrict__ grad_output_gate_normalized,
	const scalar_t* __restrict__ grad_output_gate_normalized_sum,
	const scalar_t* __restrict__ grad_output_gate_normalized_prod_sum,
	const scalar_t* __restrict__ output_gate_normalized,
	const scalar_t* __restrict__ output_gate_std,
	scalar_t* __restrict__ grad_gate_raw,
	const scalar_t* __restrict__ weight_co,
	const scalar_t* __restrict__ grad_new_cell_wrt_tanh_new_cell,
	scalar_t* __restrict__ grad_cell,
	scalar_t* __restrict__ grad_new_cell,
	const scalar_t* __restrict__ gamma_new_cell,
	const size_t batch_size,
	const size_t state_size,
	const size_t state_size_3)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int local_state_idx = batch * state_size + column;
			const int gate_idx = local_state_idx + (batch + 1) * state_size_3;
			scalar_t grad_val = (state_size * grad_output_gate_normalized[local_state_idx]
								 - grad_output_gate_normalized_sum[batch]
								 - output_gate_normalized[local_state_idx] * grad_output_gate_normalized_prod_sum[batch]) / output_gate_std[batch];
			grad_gate_raw[gate_idx] = grad_val;
			grad_val = grad_val * weight_co[column] + grad_new_cell_wrt_tanh_new_cell[local_state_idx] + grad_cell[local_state_idx];
			grad_new_cell[local_state_idx] = grad_val;
			grad_cell[local_state_idx] = grad_val * gamma_new_cell[column];
		}
	}
}

template <typename scalar_t>
__global__ void backward_loop_part_2(
	scalar_t* __restrict__ grad_cell,
	const scalar_t* __restrict__ grad_cell_sum,
	const scalar_t* __restrict__ grad_cell_prod_sum,
	const scalar_t* __restrict__ new_cell_normalized,
	const scalar_t* __restrict__ new_cell_std,
	const size_t batch_size,
	const size_t state_size)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int local_state_idx = batch * state_size + column;
			grad_cell[local_state_idx] = (state_size * grad_cell[local_state_idx]
									      - grad_cell_sum[batch]
								          - new_cell_normalized[local_state_idx] * grad_cell_prod_sum[batch]) / new_cell_std[batch];
		}
	}
}

template <typename scalar_t>
__global__ void backward_loop_part_3(
	const scalar_t* __restrict__ grad_cell,
	scalar_t* __restrict__ grad_gate_layer_normalized,
	const scalar_t* __restrict__ gamma_f,
	const scalar_t* __restrict__ gamma_i,
	const scalar_t* __restrict__ gamma_g,
	scalar_t* __restrict__ grad_fig_gate_normalized,
	const size_t batch_size,
	const size_t state_size,
	const size_t state_size_3)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int process_idx = blockIdx.z;
			if (process_idx < 3)
			{
				const int local_state_idx = batch * state_size + column;
				const int fig_idx = batch * state_size_3 + process_idx * state_size + column;
				const int gate_idx = fig_idx + batch * state_size;
				scalar_t grad_val = grad_cell[local_state_idx] * grad_gate_layer_normalized[gate_idx];
				grad_gate_layer_normalized[gate_idx] = grad_val;
				if (process_idx == 0)
				{
					grad_val *= gamma_f[column];
				}
				else{if (process_idx == 1)
				{
					grad_val *= gamma_i[column];
				}
				else
				{
					grad_val *= gamma_g[column];
				}}
				grad_fig_gate_normalized[fig_idx] = grad_val;
			}
		}
	}
}

template <typename scalar_t>
__global__ void backward_loop_part_4(
	const scalar_t* __restrict__ grad_fig_gate_normalized,
	const scalar_t* __restrict__ grad_fig_gate_normalized_sum,
	const scalar_t* __restrict__ grad_fig_gate_normalized_prod_sum,
	const scalar_t* __restrict__ gate_fig_normalized,
	const scalar_t* __restrict__ gate_fig_std,
	scalar_t* __restrict__ grad_gate_raw,
	const size_t batch_size,
	const size_t state_size,
	const size_t state_size_3)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int process_idx = blockIdx.z;
			if (process_idx < 3)
			{
				const int fig_idx = batch * state_size_3 + process_idx * state_size + column;
				const int reduced_fig_idx = batch * 3 + process_idx;
				scalar_t grad_val = (state_size * grad_fig_gate_normalized[fig_idx]
									 - grad_fig_gate_normalized_sum[reduced_fig_idx]
									 - gate_fig_normalized[fig_idx] * grad_fig_gate_normalized_prod_sum[reduced_fig_idx]) / gate_fig_std[reduced_fig_idx];
				grad_gate_raw[fig_idx + batch * state_size] = grad_val;
			}
		}
	}
}

template <typename scalar_t>
__global__ void backward_loop_part_5(
	const scalar_t* __restrict__ grad_hci,
	const scalar_t* __restrict__ forget_gate,
	scalar_t* __restrict__ grad_hidden,
	scalar_t* __restrict__ grad_cell,
	scalar_t* __restrict__ grad_input,
	const size_t batch_size,
	const size_t input_size,
	const size_t state_size,
	const size_t state_size_2,
	const size_t X_size)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < X_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < batch_size)
		{
			const int grad_idx = batch * X_size + column;
			if (column < state_size)
			{
				grad_hidden[batch * state_size + column] = grad_hci[grad_idx];
			}
			else{if (column < state_size_2)
			{
				const int local_state_idx = (batch - 1) * state_size + column;
				grad_cell[local_state_idx] = grad_hci[grad_idx] + grad_cell[local_state_idx] * forget_gate[local_state_idx];
			}
			else
			{
				grad_input[batch * input_size + column - state_size_2] = grad_hci[grad_idx];
			}}
		}
	}
}

template <typename scalar_t>
__global__ void backward_final(
	scalar_t* __restrict__ sum_to_get_grads,
	const scalar_t* __restrict__ grad_gates_raw,
	const scalar_t* __restrict__ cells,
	const scalar_t* __restrict__ grad_gates_layer_normalized,
	const scalar_t* __restrict__ gates_fig_normalized,
	const scalar_t* __restrict__ gates_o_normalized,
	const scalar_t* __restrict__ grad_new_cells,
	const scalar_t* __restrict__ new_cells_normalized,
	const size_t n_total_batches,
	const size_t batch_size,
	const size_t state_size,
	const size_t state_size_2,
	const size_t state_size_3,
	const size_t gate_size)
{
	const int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < state_size)
	{
		const int batch = blockIdx.y * blockDim.y + threadIdx.y;
		if (batch < n_total_batches)
		{
			const int process_idx = blockIdx.z;
			if (process_idx < 13)
			{
				const int store_idx = batch * (gate_size * 3 + state_size) + process_idx * state_size + column;
				/*if (process_idx == 0)
				{
					sum_to_get_grads[store_idx] = grad_gates_raw[batch * gate_size + column] * cells[batch * state_size + column];
				}
				else{
				if (process_idx == 1)
				{
					sum_to_get_grads[store_idx] = grad_gates_raw[batch * gate_size + column + state_size] * cells[batch * state_size + column];
				}
				else{
				if (process_idx == 2)
				{
					sum_to_get_grads[store_idx] = grad_gates_raw[batch * gate_size + column + state_size_3] * cells[(batch + batch_size) * state_size + column];
				}
				else{
				if (process_idx == 3)
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column];
				}
				else{
				if (process_idx == 4)
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column + state_size];
				}
				else{
				if (process_idx == 5)
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column + state_size_2];
				}
				else{
				if (process_idx == 6)
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column + state_size_3];
				}
				else{
				if (process_idx == 7)
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column] * gates_fig_normalized[batch * state_size_3 + column];
				}
				else{
				if (process_idx == 8)
				{
					const int norm_idx = batch * state_size_3 + column + state_size;
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[norm_idx + batch * state_size] * gates_fig_normalized[norm_idx];
				}
				else{
				if (process_idx == 9)
				{
					const int norm_idx = batch * state_size_3 + column + state_size_2;
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[norm_idx + batch * state_size] * gates_fig_normalized[norm_idx];
				}
				else{
				if (process_idx == 10)
				{
					const int norm_idx = batch * state_size + column;
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[norm_idx + (batch + 1) * state_size_3] * gates_o_normalized[norm_idx];
				}
				else{
				if (process_idx == 11)
				{
					sum_to_get_grads[store_idx] = grad_new_cells[batch * state_size + column] * new_cells_normalized[batch * state_size + column];
				}
				else
				{
					sum_to_get_grads[store_idx] = grad_new_cells[batch * state_size + column];
				}}}}}}}}}}}}*/
				switch (process_idx)
				{
				case 0:
				{
					sum_to_get_grads[store_idx] = grad_gates_raw[batch * gate_size + column] * cells[batch * state_size + column];
					break;
				}
				case 1:
				{
					sum_to_get_grads[store_idx] = grad_gates_raw[batch * gate_size + column + state_size] * cells[batch * state_size + column];
					break;
				}
				case 2:
				{
					sum_to_get_grads[store_idx] = grad_gates_raw[batch * gate_size + column + state_size_3] * cells[(batch + batch_size) * state_size + column];
					break;
				}
				case 3:
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column];
					break;
				}
				case 4:
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column + state_size];
					break;
				}
				case 5:
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column + state_size_2];
					break;
				}
				case 6:
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column + state_size_3];
					break;
				}
				case 7:
				{
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[batch * gate_size + column] * gates_fig_normalized[batch * state_size_3 + column];
					break;
				}
				case 8:
				{
					const int norm_idx = batch * state_size_3 + column + state_size;
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[norm_idx + batch * state_size] * gates_fig_normalized[norm_idx];
					break;
				}
				case 9:
				{
					const int norm_idx = batch * state_size_3 + column + state_size_2;
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[norm_idx + batch * state_size] * gates_fig_normalized[norm_idx];
					break;
				}
				case 10:
				{
					const int norm_idx = batch * state_size + column;
					sum_to_get_grads[store_idx] = grad_gates_layer_normalized[norm_idx + (batch + 1) * state_size_3] * gates_o_normalized[norm_idx];
					break;
				}
				case 11:
				{
					sum_to_get_grads[store_idx] = grad_new_cells[batch * state_size + column] * new_cells_normalized[batch * state_size + column];
					break;
				}
				case 12:
				{
					sum_to_get_grads[store_idx] = grad_new_cells[batch * state_size + column];
					break;
				}
				}
			}
		}
	}
}

std::vector<at::Tensor> ln_peephole_lstm_layer_cuda_backward(
	at::Tensor &grad_output,
	at::Tensor &grad_hidden,
	at::Tensor &grad_cell,
	at::Tensor const &input,
	at::Tensor const &hiddens,
	at::Tensor const &cells,
	at::Tensor const &gates_fig,
	at::Tensor const &gates_fig_normalized,
	at::Tensor &gates_fig_stds,
	at::Tensor const &gates_o,
	at::Tensor const &gates_o_normalized,
	at::Tensor &gates_o_stds,
	at::Tensor const &new_cells_normalized,
	at::Tensor &new_cells_stds,
	at::Tensor &tanh_new_cells,
	at::Tensor const &dropout,
	at::Tensor const &weight_ih,
	at::Tensor const &weight_hh,
	at::Tensor const &weight_ch,
	at::Tensor const &gamma_f,
	at::Tensor const &gamma_i,
	at::Tensor const &gamma_g,
	at::Tensor const &gamma_o,
	at::Tensor const &gamma_new_cell)
{
	const auto sequence_length = input.size(0);
	const auto batch_size = input.size(1);
	const auto state_size = hiddens.size(2);
	const auto state_size_2 = state_size + state_size;
	const auto state_size_3 = state_size_2 + state_size;
	const auto gate_size = state_size_3 + state_size;
	const auto input_size = input.size(2);
	const auto X_size = input_size + state_size_2;

	const auto n_total_batches = batch_size * sequence_length;

	const auto dropout_candidate_cell = dropout[0];
	const auto dropout_output = dropout[1];

	const auto weights = at::cat({ weight_hh,
								   at::cat({ weight_ch.slice(0, 0, state_size).diag(),
											 weight_ch.slice(0, state_size, state_size_2).diag(),
											 at::zeros({ state_size_2, state_size }, weight_ch.options()) }),
								   weight_ih }, 1);
	const auto weight_co = weight_ch.slice(0, state_size_2);

	auto grad_input = at::empty_like(input);

	auto grad_gates_layer_normalized = at::empty({ sequence_length, batch_size, gate_size }, gates_fig.options());
	auto grad_gates_raw = at::empty_like(grad_gates_layer_normalized);
	auto grad_new_cells = at::empty_like(tanh_new_cells);
	auto grad_new_cells_wrt_tanh_new_cell = at::empty_like(tanh_new_cells);

	auto grad_output_gate_normalized = at::empty({ batch_size, state_size }, grad_gates_raw.options());
	auto grad_fig_gate_normalized = at::empty({ batch_size, 3, state_size }, grad_gates_raw.options());
	auto forget_gates = at::empty_like(tanh_new_cells);

	at::Tensor sum_to_get_grads;

	const dim3 threads(64, 8);
	const dim3 blocks_0((state_size + threads.x - 1) / threads.x,
						(n_total_batches + threads.y - 1) / threads.y,
						6);
	const dim3 blocks_1((state_size + threads.x - 1) / threads.x,
						(batch_size + threads.y - 1) / threads.y);
	const dim3 blocks_2((state_size + threads.x - 1) / threads.x,
						(batch_size + threads.y - 1) / threads.y,
						3);
	const dim3 blocks_3((X_size + threads.x - 1) / threads.x,
						(batch_size + threads.y - 1) / threads.y);
	const dim3 blocks_4((state_size + threads.x - 1) / threads.x,
						(n_total_batches + threads.y - 1) / threads.y,
						13);

	AT_DISPATCH_FLOATING_TYPES(gates_fig.type(), "ln_peephole_lstm_layer_cuda_backward", ([&] {
		backward_preparation<scalar_t> <<<blocks_0, threads>>> (
			grad_output.data<scalar_t>(),
			dropout_output.data<scalar_t>(),
			dropout_candidate_cell.data<scalar_t>(),
			cells.data<scalar_t>(),
			gates_fig.data<scalar_t>(),
			gates_o.data<scalar_t>(),
			grad_gates_layer_normalized.data<scalar_t>(),
			gates_fig_stds.data<scalar_t>(),
			gates_o_stds.data<scalar_t>(),
			new_cells_stds.data<scalar_t>(),
			tanh_new_cells.data<scalar_t>(),
			grad_new_cells_wrt_tanh_new_cell.data<scalar_t>(),
			forget_gates.data<scalar_t>(),
			n_total_batches,
			state_size,
			state_size_2,
			state_size_3,
			gate_size);

		for (int i = sequence_length - 1; i >= 0; i--)
		{
			backward_loop_part_0<scalar_t> <<<blocks_1, threads>>> (
				grad_hidden.data<scalar_t>(),
				grad_new_cells_wrt_tanh_new_cell[i].data<scalar_t>(),
				grad_output[i].data<scalar_t>(),
				grad_gates_layer_normalized[i].data<scalar_t>(),
				gamma_o.data<scalar_t>(),
				grad_output_gate_normalized.data<scalar_t>(),
				batch_size,
				state_size,
				state_size_3);

			backward_loop_part_1<scalar_t> <<<blocks_1, threads>>> (
				grad_output_gate_normalized.data<scalar_t>(),
				grad_output_gate_normalized.sum(/*dim=*/1, /*keepdim=*/false).data<scalar_t>(),
				grad_output_gate_normalized.mul(gates_o_normalized[i]).sum(/*dim=*/1, /*keepdim=*/false).data<scalar_t>(),
				gates_o_normalized[i].data<scalar_t>(),
				gates_o_stds[i].data<scalar_t>(),
				grad_gates_raw[i].data<scalar_t>(),
				weight_co.data<scalar_t>(),
				grad_new_cells_wrt_tanh_new_cell[i].data<scalar_t>(),
				grad_cell.data<scalar_t>(),
				grad_new_cells[i].data<scalar_t>(),
				gamma_new_cell.data<scalar_t>(),
				batch_size,
				state_size,
				state_size_3);

			backward_loop_part_2<scalar_t> <<<blocks_1, threads>>> (
				grad_cell.data<scalar_t>(),
				grad_cell.sum(/*dim=*/1, /*keepdim=*/false).data<scalar_t>(),
				grad_cell.mul(new_cells_normalized[i]).sum(/*dim=*/1, /*keepdim=*/false).data<scalar_t>(),
				new_cells_normalized[i].data<scalar_t>(),
				new_cells_stds[i].data<scalar_t>(),
				batch_size,
				state_size);

			backward_loop_part_3<scalar_t> <<<blocks_2, threads>>> (
				grad_cell.data<scalar_t>(),
				grad_gates_layer_normalized[i].data<scalar_t>(),
				gamma_f.data<scalar_t>(),
				gamma_i.data<scalar_t>(),
				gamma_g.data<scalar_t>(),
				grad_fig_gate_normalized.data<scalar_t>(),
				batch_size,
				state_size,
				state_size_3);

			backward_loop_part_4<scalar_t> <<<blocks_2, threads>>> (
				grad_fig_gate_normalized.data<scalar_t>(),
				grad_fig_gate_normalized.sum(/*dim=*/2, /*keepdim=*/false).data<scalar_t>(),
				grad_fig_gate_normalized.mul(gates_fig_normalized[i]).sum(/*dim=*/2, /*keepdim=*/false).data<scalar_t>(),
				gates_fig_normalized[i].data<scalar_t>(),
				gates_fig_stds[i].data<scalar_t>(),
				grad_gates_raw[i].data<scalar_t>(),
				batch_size,
				state_size,
				state_size_3);

			backward_loop_part_5<scalar_t> <<<blocks_3, threads>>> (
				grad_gates_raw[i].mm(weights).data<scalar_t>(),
				forget_gates[i].data<scalar_t>(),
				grad_hidden.data<scalar_t>(),
				grad_cell.data<scalar_t>(),
				grad_input[i].data<scalar_t>(),
				batch_size,
				input_size,
				state_size,
				state_size_2,
				X_size);
		}
		sum_to_get_grads = at::empty({ sequence_length * batch_size, gate_size * 3 + state_size }, weights.options());
		backward_final<scalar_t> <<<blocks_4, threads>>> (
			sum_to_get_grads.data<scalar_t>(),
			grad_gates_raw.data<scalar_t>(),
			cells.data<scalar_t>(),
			grad_gates_layer_normalized.data<scalar_t>(),
			gates_fig_normalized.data<scalar_t>(),
			gates_o_normalized.data<scalar_t>(),
			grad_new_cells.data<scalar_t>(),
			new_cells_normalized.data<scalar_t>(),
			n_total_batches,
			batch_size,
			state_size,
			state_size_2,
			state_size_3,
			gate_size);
	}));
	const auto grad_weight_ih_hh = grad_gates_raw.view({ sequence_length * batch_size, gate_size }).t().mm(at::cat({ input, hiddens }, 2).view({ sequence_length * batch_size, input_size + state_size }));

	const auto bunch_of_grads = sum_to_get_grads.sum(/*dim=*/0, /*keepdim=*/false);

	return { grad_input,
			 grad_weight_ih_hh.slice(1, 0, input_size).contiguous(),
			 grad_weight_ih_hh.slice(1, input_size).contiguous(),
			 bunch_of_grads.slice(0, 0, state_size_3),
			 bunch_of_grads.slice(0, state_size_3, gate_size + state_size_3),
			 bunch_of_grads.slice(0, gate_size + state_size_3, gate_size * 2),
			 bunch_of_grads.slice(0, gate_size * 2, gate_size * 2 + state_size),
			 bunch_of_grads.slice(0, gate_size * 2 + state_size, gate_size * 2 + state_size_2),
			 bunch_of_grads.slice(0, gate_size * 2 + state_size_2, gate_size * 2 + state_size_3),
			 bunch_of_grads.slice(0, gate_size * 2 + state_size_3, gate_size * 3),
			 bunch_of_grads.slice(0, gate_size * 3),
			 grad_hidden,
			 grad_cell };
}
